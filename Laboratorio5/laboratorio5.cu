#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <opencv2/opencv.hpp>

#define BLUR_SIZE 10

// Kernel para convertir la imagen a escala de grises
__global__ void rgbToGray(unsigned char* inputImage, unsigned char* outputImage, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int grayOffset = y * width + x;
        int rgbOffset = grayOffset * 3;

        unsigned char r = inputImage[rgbOffset];
        unsigned char g = inputImage[rgbOffset + 1];
        unsigned char b = inputImage[rgbOffset + 2];

        outputImage[grayOffset] = 0.21f * r + 0.72f * g + 0.07f * b;
    }
}

// Kernel para aplicar un filtro de desenfoque
__global__
void blurKernel(unsigned char* in, unsigned char* out, int w, int h) {
    int Col = blockIdx.x * blockDim.x + threadIdx.x;
    int Row = blockIdx.y * blockDim.y + threadIdx.y;

    if (Col < w && Row < h) {
        int pixVal = 0;
        int pixels = 0;

        // Obtener el promedio del cuadro BLUR_SIZE x BLUR_SIZE
        for (int blurRow = -BLUR_SIZE; blurRow < BLUR_SIZE + 1; ++blurRow) {
            for (int blurCol = -BLUR_SIZE; blurCol < BLUR_SIZE + 1; ++blurCol) {
                int curRow = Row + blurRow;
                int curCol = Col + blurCol;
                // Verificar que tengamos un píxel de imagen válido
                if (curRow > -1 && curRow < h && curCol > -1 && curCol < w) {
                    pixVal += in[curRow * w + curCol];
                    pixels++; // Mantener un registro del número de píxeles en el promedio
                }
            }
        }
        // Escribir nuestro nuevo valor de píxel
        out[Row * w + Col] = (unsigned char)(pixVal / pixels);
    }
}

int main() {
    std::string inputImagePath = "C:/Users/alvaro/Documents/Capturas/input.jpg";
    std::string outputImagePathGray = "C:/Users/alvaro/Documents/Capturas/output_gray.jpg";
    std::string outputImagePathBlur = "C:/Users/alvaro/Documents/Capturas/output_blur.jpg";

    // Cargar la imagen de entrada
    cv::Mat inputImage = cv::imread(inputImagePath);
    if (inputImage.empty()) {
        std::cerr << "Error: Couldn't load the image." << std::endl;
        return -1;
    }

    int width = inputImage.cols;
    int height = inputImage.rows;

    size_t imageSize = width * height * inputImage.channels();
    size_t grayImageSize = width * height * sizeof(unsigned char);

    unsigned char* devInputImage, * devOutputImageGray, * devOutputImageBlur;
    hipMalloc((void**)&devInputImage, imageSize);
    hipMalloc((void**)&devOutputImageGray, grayImageSize);
    hipMalloc((void**)&devOutputImageBlur, grayImageSize);

    hipMemcpy(devInputImage, inputImage.data, imageSize, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((width + threadsPerBlock.x - 1) / threadsPerBlock.x,
        (height + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Convertir a escala de grises
    rgbToGray << <numBlocks, threadsPerBlock >> > (devInputImage, devOutputImageGray, width, height);

    // Aplicar el filtro de desenfoque
    blurKernel << <numBlocks, threadsPerBlock >> > (devOutputImageGray, devOutputImageBlur, width, height);

    unsigned char* outputImageGray = new unsigned char[grayImageSize];
    unsigned char* outputImageBlur = new unsigned char[grayImageSize];

    // Copiar los resultados de la GPU a la memoria del host
    hipMemcpy(outputImageGray, devOutputImageGray, grayImageSize, hipMemcpyDeviceToHost);
    hipMemcpy(outputImageBlur, devOutputImageBlur, grayImageSize, hipMemcpyDeviceToHost);

    // Crear imágenes Mat a partir de los datos obtenidos
    cv::Mat grayImage(height, width, CV_8UC1, outputImageGray);
    cv::Mat blurImage(height, width, CV_8UC1, outputImageBlur);

    // Mostrar las imágenes procesadas
    cv::imshow("Original Image", inputImage);
    cv::imshow("Blurred Image", blurImage);
    cv::imshow("Gray Image", grayImage);

    cv::waitKey(0);

    // Guardar las imágenes en archivos de salida
    cv::imwrite(outputImagePathGray, grayImage);
    cv::imwrite(outputImagePathBlur, blurImage);

    // Liberar la memoria
    hipFree(devInputImage);
    hipFree(devOutputImageGray);
    hipFree(devOutputImageBlur);
    delete[] outputImageGray;
    delete[] outputImageBlur;

    return 0;
}
